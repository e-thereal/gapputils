#include "hip/hip_runtime.h"
#include "NLML.h"

#include <thrust/copy.h>
#include <cassert>
#include <hipblas.h>
#include <fstream>
#include <iostream>
#include <sstream>

#include <vector>

#include "gpgpu.h"

using namespace std;

namespace GaussianProcesses {

NLML::NLML(float* x, float *y, int n, int d)
 : n(n), d(d), d_alpha(n), d_diag(n),
   d_x(x, x + (n * d)), d_y(y, y + n), d_length(d), length(d)
{
  bn = n;
  if (n % BLOCK_SIZE) {
    bn += BLOCK_SIZE - (n % BLOCK_SIZE);
  }
  d_K.resize(bn * bn);
}

NLML::~NLML(void)
{
}

double NLML::eval(const DomainType& parameter) {

  assert(parameter.size() == 2 + d);
  const float sigmaF = exp(parameter[0]);
  const float sigmaN = exp(parameter[1]);
  
  for (int i = 0; i < d; ++i)
    length[i] = exp(parameter[i + 2]);

  return eval(sigmaF, sigmaN, &length[0]);
}

NLML::DomainType NLML::gradient(const DomainType& parameter) {
  assert(parameter.size() == 2 + d);
  const float sigmaF = exp(parameter[0]);
  const float sigmaN = exp(parameter[1]);

  for (int i = 0; i < d; ++i)
    length[i] = exp(parameter[i + 2]);

  return gradient(sigmaF, sigmaN, &length[0]);
}

// #define PRINT_MATRIX

#ifdef PRINT_MATRIX
#define PRINTMATRIX(a, b, c, d, e, f) printMatrix(a, b, c, d, e, f)
#else
#define PRINTMATRIX(a, b, c, d, e, f)
#endif

double NLML::eval(float sigmaF, float sigmaN, float* length) {
  const float logTwoPiHalf = 0.918938533f;
#ifdef PRINT_MATRIX
  std::ofstream outfile("nlmlgpu.txt");
#endif
  thrust::copy(d_y.begin(), d_y.end(), d_alpha.begin());
  thrust::copy(length, length + d, d_length.begin());
  covSEFast(n, n, d, d_K.data().get(), bn, d_x.data().get(), n, d_x.data().get(), n, sigmaF, sigmaN, d_length.data().get());

  PRINTMATRIX(outfile, "Kfast", d_K.data().get(), n, n, bn);

  //covSE(n, n, d, d_K.data().get(), bn, d_x.data().get(), n, d_x.data().get(), n, sigmaF, sigmaN, d_length.data().get());
  PRINTMATRIX(outfile, "K", d_K.data().get(), n, n, bn);
  PRINTMATRIX(outfile, "x", d_x.data().get(), n, d, n);
  PRINTMATRIX(outfile, "y", d_alpha.data().get(), n, 1, n);

  cholesky_cuda_block(d_K.data().get(), bn);
  PRINTMATRIX(outfile, "U", d_K.data().get(), n, n, bn);

  hipblasStrsv('U', 'T', 'N', n, d_K.data().get(), bn, d_alpha.data().get(), 1);
  PRINTMATRIX(outfile, "alpha", d_alpha.data().get(), n, 1, n);

  // 0.5 * alpha' * alpha + log(det(K)) + n/2 * ln(2 PI)
  float l2 = l2norm(d_alpha.data().get(), n);
  float logDet = Strldet(d_K.data().get(), n, bn);
  //cout << "L2: " << l2 << endl;
  //cout << "logDet: " << logDet << endl;

  float ret = 0.5f * l2 + logDet + n * logTwoPiHalf;
#ifdef PRINT_MATRIX
  outfile << "nlml = " << ret << std::endl;
  outfile.close();
#endif
  return ret;
}

NLML::DomainType NLML::gradient(float sigmaF, float sigmaN, float* length) {
  vector<double> gradient(d + 2);
#ifdef PRINT_MATRIX
  ofstream outfile("dnlml.txt");
#endif

  thrust::copy(d_y.begin(), d_y.end(), d_alpha.begin());
  PRINTMATRIX(outfile, "y", d_alpha.data().get(), n, 1, n);
  thrust::copy(length, length + d, d_length.begin());
  covSEFast(n, n, d, d_K.data().get(), bn, d_x.data().get(), n, d_x.data().get(), n, sigmaF, sigmaN, d_length.data().get());
  PRINTMATRIX(outfile, "K", d_K.data().get(), n, n, bn);

  cholesky_cuda_block(d_K.data().get(), bn);
  PRINTMATRIX(outfile, "L", d_K.data().get(), n, n, bn);

  // alpha = L'\(L\y) = K^{-1}y
  hipblasStrsv('U', 'T', 'N', n, d_K.data().get(), bn, d_alpha.data().get(), 1);
  hipblasStrsv('U', 'N', 'N', n, d_K.data().get(), bn, d_alpha.data().get(), 1);
  PRINTMATRIX(outfile, "alpha", d_alpha.data().get(), n, 1, n);

  // d_W = invK = L'\(L\I) (I = identity)
  thrust::device_vector<float> d_W(n * n);
  setToIdentity(d_W.data().get(), n);
  PRINTMATRIX(outfile, "I", d_W.data().get(), n, n, n);
  hipblasStrsm('L', 'U', 'T', 'N', n, n, 1.f, d_K.data().get(), bn, d_W.data().get(), n);
  hipblasStrsm('L', 'U', 'N', 'N', n, n, 1.f, d_K.data().get(), bn, d_W.data().get(), n);
  PRINTMATRIX(outfile, "invK", d_W.data().get(), n, n, n);

  // W = invK - alpha * alpha' = -1.0 * alpha * alpha' + 1.0 * invK
  hipblasSgemm('N', 'T', n, n, 1, -1.f, d_alpha.data().get(), n, d_alpha.data().get(), n,
      1.0f, d_W.data().get(), n);
  PRINTMATRIX(outfile, "W", d_W.data().get(), n, n, n);

  thrust::device_vector<float> d_dK(n * n);
  for (int iparam = 0; iparam < gradient.size(); ++iparam) {
    stringstream sname;
    sname << "dK_" << iparam;

    // dK/dO_i
    derivSE(n, n, d, d_dK.data().get(), n, d_x.data().get(), n, d_x.data().get(), n, sigmaF, sigmaN, d_length.data().get(), iparam);
    PRINTMATRIX(outfile, sname.str().c_str(), d_dK.data().get(), n, n, n);

    // dnlml_i = 1/2 * tr(W * dK/dO_i) = 1/2 * sum(W .* dK/dO_i)
    gradient[iparam] = 0.5 * thrust::inner_product(d_W.begin(), d_W.end(), d_dK.begin(), 0.0f);
#ifdef PRINT_MATRIX
    outfile << "g_" << iparam << " = " << gradient[iparam] << ";" << endl;
#endif
  }
#ifdef PRINT_MATRIX
  outfile.close();
#endif

  return gradient;
}

}
