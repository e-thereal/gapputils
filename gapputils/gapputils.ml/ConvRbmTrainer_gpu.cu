#include "hip/hip_runtime.h"
/*
 * ConvRbmTrainer_gpu.cu
 *
 *  Created on: Mar 5, 2012
 *      Author: tombr
 */

#define BOOST_TYPEOF_COMPLIANT
#include "ConvRbmTrainer.h"

#include <iostream>

#include <capputils/Verifier.h>

#include <boost/timer.hpp>
#include <tbblas/tensor_proxy.hpp>

//#include "sampling.hpp"

#include "RbmModel.h"

#include <hiprand.h>

namespace gapputils {

namespace ml {

#define LOCATE(a,b) std::cout << #b": " << (char*)&a._##b - (char*)&a << std::endl

template<class T>
struct softmax : thrust::binary_function<T, unsigned, T> {

  softmax(unsigned width, unsigned blockSize) : width(width), blockSize(blockSize) { }

  __host__ __device__
  T operator()(const T& value, const unsigned& idx) const {
    T res = 0;
    const int offset = (idx % width) % blockSize + ((idx / width) % blockSize) * width;
    for (unsigned j = 0; j < blockSize; ++j)
      for (unsigned i = 0; i < blockSize; ++i)
        res += exp(*(&value + i + j * width - offset));
    return exp(value) / (1 + res);
  }

private:
  unsigned blockSize, width;
};

void ConvRbmTrainer::execute(gapputils::workflow::IProgressMonitor* monitor) const {
  using namespace thrust::placeholders;

  boost::timer timer;

  typedef tbblas::tensor_proxy<device_tensor_t::iterator, 3> device_proxy_t;

  if (!data)
    data = new ConvRbmTrainer();

  //  std::cout << "Device:" << std::endl;
  //  ConvRbmTrainer test;
  //  LOCATE(test, InitialModel);
  //  LOCATE(test, Tensors);
  //  LOCATE(test, SampleVisibles);
  //  LOCATE(test, EpochCount);
  //  LOCATE(test, BatchSize);
  //  LOCATE(test, LearningRate);
  //  LOCATE(test, Model);

  if (!capputils::Verifier::Valid(*this))
    return;

  if (!getInitialModel()) {
    std::cout << "[Warning] No initial model given. Aborting!" << std::endl;
    return;
  }

  if (!getTensors() || getTensors()->size() == 0) {
    std::cout << "[Warning] No training data given. Aborting!" << std::endl;
    return;
  }

  std::cout << "Building ConvRBM ..." << std::endl;

  hiprandGenerator_t gen;
  hiprandStatus_t status;
  if ((status = hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT)) != HIPRAND_STATUS_SUCCESS) {
    std::cout << "[Warning] Could not create random number generator: " << status << std::endl;
    return;
  }

  // Calculate the mean and the std of all features
  const unsigned sampleCount = getTensors()->size();
  const int batchSize = getBatchSize();

  boost::shared_ptr<ConvRbmModel> crbm = getInitialModel()->clone();

  if (crbm->getIsGaussian()) {
    assert(0); // Gaussian case not implemented yet.
  }

  const unsigned dimCount = ConvRbmModel::dimCount;
  const unsigned filterCount = crbm->getFilters()->size();
  const unsigned blockSize = crbm->getPoolingBlockSize();
  const host_tensor_t::dim_t& filterDim = crbm->getFilters()->at(0)->size();
  const host_tensor_t::dim_t& inputDim = getTensors()->at(0)->size();
  host_tensor_t::dim_t layerDim, paddedDim, start;

  int filterWeightCount = 1, layerVoxelCount = 1, inputVoxelCount = 1;
  for (unsigned i = 0; i < dimCount; ++i) {
    layerDim[i] = inputDim[i] - filterDim[i] + 1;
    paddedDim[i] = inputDim[i] + filterDim[i] - 1;
    start[i] = filterDim[i] - 1;
    filterWeightCount *= filterDim[i];
    layerVoxelCount *= layerDim[i];
    inputVoxelCount *= inputDim[i];
  }

  assert((layerDim[0] % blockSize) == 0);
  assert((layerDim[1] % blockSize) == 0);
  assert((layerVoxelCount % 2) == 0);
  assert((inputVoxelCount % 2) == 0);

  // Train the RBM
  std::vector<boost::shared_ptr<host_tensor_t> >& X = *getTensors();

  // Copy filters to the device
  std::vector<boost::shared_ptr<host_tensor_t> >& filters = *crbm->getFilters();
  std::vector<device_tensor_t > F;
  for (unsigned i = 0; i < filters.size(); ++i) {
    device_tensor_t filter(filters[i]->size());
    thrust::copy(filters[i]->begin(), filters[i]->end(), filter.begin());
    F.push_back(filter);
  }

  value_t b = crbm->getVisibleBias();
  std::vector<value_t>& c = *crbm->getHiddenBiases();

  std::cout << "[Info] ConvRBM initialized: " << timer.elapsed() << " s" << std::endl;

  // Start the learning
  const int batchCount = sampleCount / batchSize;
  value_t epsilonw =  getLearningRate();      // Learning rate for weights
  value_t epsilonvb = getLearningRate();      // Learning rate for biases of visible units
  value_t epsilonhb = getLearningRate();      // Learning rate for biases of hidden units
  value_t weightcost = 0; // 0.0002;
  value_t initialmomentum = 0.5; //65; // 0.5f;
  value_t finalmomentum = 0.9; // 65; // 0.9f;
  value_t momentum;

  device_tensor_t v(inputDim), vneg(inputDim), vtemp(inputDim), padded(paddedDim);
  thrust::fill(padded.begin(), padded.end(), value_t(0));
  std::vector<device_tensor_t> poshidprobs, poshidstates, posvishid, neghidprobs, neghidstates, negvishid, Finc, Fincbatch;
  for (unsigned i = 0; i < filterCount; ++i) {
    poshidprobs.push_back(device_tensor_t(layerDim));
    poshidstates.push_back(device_tensor_t(layerDim));
    posvishid.push_back(device_tensor_t(filterDim));

    neghidprobs.push_back(device_tensor_t(layerDim));
    neghidstates.push_back(device_tensor_t(layerDim));
    negvishid.push_back(device_tensor_t(filterDim));
    Finc.push_back(device_tensor_t(filterDim));
    thrust::fill(Finc[i].begin(), Finc[i].end(), value_t(0));
    Fincbatch.push_back(device_tensor_t(filterDim));
  }
  value_t posvisact, negvisact, binc = 0, bincbatch;
  std::vector<value_t> poshidact(filterCount), neghidact(filterCount),
      cinc(filterCount, 0), cincbatch(filterCount, 0),
      cspa(filterCount, 0), cspabatch(filterCount, 0);

  const int epochCount = getEpochCount();

  std::cout << "[Info] Preparation finished after " << timer.elapsed() << " s" << std::endl;
  std::cout << "[Info] Starting training" << std::endl;
  timer.restart();
  const int wCount = filterWeightCount * filterCount;
  boost::shared_ptr<std::vector<float> > vFilters(new std::vector<float>(wCount));
  data->setFilters(vFilters);
  for (int iEpoch = 0; iEpoch < epochCount && (monitor ? !monitor->getAbortRequested() : true); ++iEpoch) {

    double error = 0;
    for (int iBatch = 0; iBatch < batchCount && (monitor ? !monitor->getAbortRequested() : true); ++iBatch) {

      for (unsigned k = 0; k < filterCount; ++k) {
        thrust::fill(Fincbatch[k].begin(), Fincbatch[k].end(), value_t(0));
        cincbatch[k] = 0;
        cspabatch[k] = 0;
      }
      bincbatch = 0;

      for (int iSample = 0; iSample < batchSize; ++iSample) {

        /*** START POSITIVE PHASE ***/
        const int randomSample = rand() % sampleCount;

        // Get current sample
        if (getUseRandomSamples())
          thrust::copy(X[randomSample]->begin(), X[randomSample]->end(), v.begin());
        else
          thrust::copy(X[iSample + iBatch * batchSize]->begin(), X[iSample + iBatch * batchSize]->end(), v.begin());

        // For each filter (Could be written as a single 4D convolution in case of a 2D image and 3D filter))
        for (unsigned k = 0; k < filterCount; ++k) {

          // Calculate p(h_k | v, F) = sigm((~F_k * v) + c_k)
          poshidstates[k] = tbblas::conv(tbblas::flip(F[k]), v, (k ? tbblas::ReuseFT2 : tbblas::ReuseFTNone));
          poshidstates[k] += c[k];               // x = ~F_k * v + c_k

          // I'm using the state array here for the sum. Not nice but works fine and saves some space
          thrust::transform(poshidstates[k].data().begin(), poshidstates[k].data().end(),
              thrust::make_counting_iterator(0), poshidprobs[k].data().begin(),
              softmax<value_t>(layerDim[0], blockSize));

//          thrust::transform(poshidprobs[k].data().begin(), poshidprobs[k].data().end(), // x = sigm(x)
//              poshidprobs[k].data().begin(), sigmoid<value_t>());

          // Calculate energy and the total activation of the hidden units
          posvishid[k] = tbblas::conv(tbblas::flip(poshidprobs[k]), v, tbblas::ReuseFT2);     // ~h_k * v
          poshidact[k] = tbblas::sum(poshidprobs[k]);

          if (iEpoch || !getCalculateBaseline())
            cspabatch[k] += getSparsityTarget() - tbblas::sum(poshidprobs[k]) / poshidprobs[k].data().size();

          // fill states with random numbers which are then used to sample the units
          // TODO: use hiprandGenerateUniform if value_t == float
          if ((status = hiprandGenerateUniformDouble(gen,
              poshidstates[k].data().data().get(),
              poshidstates[k].data().size())) != HIPRAND_STATUS_SUCCESS)
          {
            std::cout << "[Error] Could not generate random numbers: " << status << std::endl;
            return;
          }

          // Sample the hidden states
          thrust::transform(
              poshidprobs[k].data().begin(), poshidprobs[k].data().end(), poshidstates[k].data().begin(),
              poshidstates[k].data().begin(), _1 > _2
          );
        }

        // Calculate the total activation of the visible units
        posvisact = tbblas::sum(v);

        /*** END OF POSITIVE PHASE ***/

        /*** START NEGATIVE PHASE ***/

        // Calculate p(v | H, F) = sigm(sum(W_k * h_k) + b)
        thrust::fill(vneg.data().begin(), vneg.data().end(), value_t(0));
        for (unsigned k = 0; k < filterCount; ++k) {
          device_proxy_t paddedProxy = tbblas::subrange(padded, start, layerDim);
          thrust::copy(poshidstates[k].begin(), poshidstates[k].end(), paddedProxy.begin());
          vtemp = tbblas::conv(F[k], padded);
          vneg += vtemp;
        }
        vneg += b;

        // For the binary case
        if (!crbm->getIsGaussian()) {
          thrust::transform(vneg.begin(), vneg.end(), vneg.begin(),
              sigmoid<value_t>());

          if (getSampleVisibles()) {
            if ((status = hiprandGenerateUniformDouble(gen, vtemp.data().data().get(), vtemp.data().size())) != HIPRAND_STATUS_SUCCESS)
            {
              std::cout << "[Error] Could not generate random numbers: " << status << std::endl;
              return;
            }

            thrust::transform(
                vneg.data().begin(), vneg.data().end(), vtemp.data().begin(),
                vneg.data().begin(), _1 > _2
            );
          }
        } else {
          if (getSampleVisibles()) {
            if ((status = hiprandGenerateNormalDouble(gen,
                vtemp.data().data().get(),
                vtemp.data().size(),
                0, 1.0)) != HIPRAND_STATUS_SUCCESS)
            {
              std::cout << "[Error] Could not generate random numbers: " << status << std::endl;
              return;
            }

            thrust::transform(
                vneg.data().begin(), vneg.data().end(), vtemp.data().begin(),
                vneg.data().begin(), thrust::plus<value_t>()
            );
          }
        }

        for (unsigned k = 0; k < filterCount; ++k) {

          // Calculate p(h_k | vneg, F) = sigm((~F_k * v) + c_k)
          neghidstates[k] = tbblas::conv(tbblas::flip(F[k]), vneg,
              (k ? tbblas::ReuseFT2 : tbblas::ReuseFTNone));               // x = ~F_k * v + c_k
          neghidstates[k] += c[k];

          thrust::transform(neghidstates[k].data().begin(), neghidstates[k].data().end(),
              thrust::make_counting_iterator(0), neghidprobs[k].data().begin(),
              softmax<value_t>(layerDim[0], blockSize));

//          thrust::transform(neghidprobs[k].data().begin(), neghidprobs[k].data().end(), // x = sigm(x)
//              neghidprobs[k].data().begin(), sigmoid<value_t>());

          // Calculate energy and the total activation of the hidden units
          negvishid[k] = tbblas::conv(tbblas::flip(neghidprobs[k]), vneg, tbblas::ReuseFT2);     // ~h_k * v
          neghidact[k] = tbblas::sum(neghidprobs[k]);
        }

        // Calculate the total activation of the visible units
        negvisact = tbblas::sum(vneg);

        /*** END OF NEGATIVE PHASE ***/

        error += thrust::inner_product(vneg.begin(), vneg.end(), v.begin(), value_t(0),
            thrust::plus<value_t>(), (_1 - _2) * (_1 - _2));
        momentum = (iEpoch > 5 ? finalmomentum : initialmomentum);

        /*** UPDATE WEIGHTS AND BIASES ***/
        if (iEpoch || !getCalculateBaseline()) {
          for (unsigned k = 0; k < filterCount; ++k) {
            Fincbatch[k] += (posvishid[k] += (-1.0 * negvishid[k]));
            cincbatch[k] += (poshidact[k] - neghidact[k]);
          }
          bincbatch = posvisact - negvisact;
        }
      }
      for (unsigned k = 0; k < filterCount; ++k) {
        Finc[k] = momentum * Finc[k] + (epsilonw / batchSize) * Fincbatch[k];
        cinc[k] = momentum * cinc[k] + (epsilonhb / batchSize) * cincbatch[k]
                  + getSparsityPenalty() * cspabatch[k] / batchSize;

        F[k] += Finc[k];
        c[k] += cinc[k];
      }
      binc = momentum * binc + (epsilonvb / batchSize) * bincbatch;
      b += binc;

      /*** END OF UPDATES ***/

      if (monitor)
        monitor->reportProgress(100. * (iEpoch * batchCount + (iBatch + 1)) / (epochCount * batchCount));
    }
    int eta = timer.elapsed() / (iEpoch + 1) * (epochCount - iEpoch - 1);
    int sec = eta % 60;
    int minutes = (eta / 60) % 60;
    int hours = eta / 3600;
    std::cout << "Epoch " << iEpoch << " error " << (error / sampleCount) << " after " << timer.elapsed() << "s. ETA: "
        << hours << " h " << minutes << " min " << sec << " s" << std::endl;

    for (unsigned i = 0; i < filterCount; ++i) {
      thrust::copy(F[i].begin(), F[i].end(), vFilters->begin() + i * filterWeightCount);
    }

    if (monitor)
      monitor->reportProgress(100. * (iEpoch + 1) / epochCount, true);
  }

  if ((status = hiprandDestroyGenerator(gen)) != HIPRAND_STATUS_SUCCESS)
  {
    std::cout << "[Error] Could not destroy random number generator: " << status << std::endl;
    return;
  }

  thrust::device_vector<value_t> temp(filterWeightCount);

  for (unsigned i = 0; i < filterCount; ++i) {
    thrust::copy(F[i].begin(), F[i].end(), vFilters->begin() + i * filterWeightCount);
//    thrust::transform(F[i].begin(), F[i].end(),
//        thrust::make_counting_iterator(0),
//        temp.begin(),
//        block_sum<value_t>(filterDim[0], 4));
//
//    thrust::copy(temp.begin(), temp.end(), vFilters->begin() + i * filterWeightCount);
  }

  data->setModel(crbm);
}

}

}


