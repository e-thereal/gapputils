/*
 * CudaBugTest_gpu.cu
 *
 *  Created on: Mar 8, 2013
 *      Author: tombr
 */

#include "CudaBugTest.h"

#include <hip/hip_runtime.h>

namespace gml {

namespace convrbm4d {

void CudaBugTest::update(IProgressMonitor* monitor) const {
  double* ptr;
  hipMalloc(&ptr, 8);
  sleep(1);
  hipFree(ptr);
//  hipDeviceReset();
}

}

}
