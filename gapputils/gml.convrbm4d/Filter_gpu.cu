#include "hip/hip_runtime.h"
/*
 * Filter_gpu.cu
 *
 *  Created on: Nov 23, 2012
 *      Author: tombr
 */

#include "Filter.h"

#include <tbblas/fft.hpp>
#include <tbblas/math.hpp>
#include <tbblas/zeros.hpp>
#include <tbblas/repeat.hpp>

#include <omp.h>

#include "math.hpp"

namespace gml {

namespace convrbm4d {

FilterChecker::FilterChecker() {
  Filter filter;
  filter.initializeClass();
  CHECK_MEMORY_LAYOUT2(Model, filter);
  CHECK_MEMORY_LAYOUT2(Inputs, filter);
  CHECK_MEMORY_LAYOUT2(Direction, filter);
  CHECK_MEMORY_LAYOUT2(GpuCount, filter);

  CHECK_MEMORY_LAYOUT2(Outputs, filter);
}

unsigned int upper_power_of_two(unsigned int v);

void Filter::update(IProgressMonitor* monitor) const {
  using namespace tbblas;

  Logbook& dlog = getLogbook();

  const unsigned dimCount = Model::dimCount;
  typedef complex<value_t> complex_t;
  typedef fft_plan<dimCount> plan_t;
  typedef tensor<value_t, dimCount, true> tensor_t;
  typedef tensor<complex_t, dimCount, true> ctensor_t;
  typedef tensor<complex_t, dimCount, false> host_ctensor_t;
  typedef tensor_t::dim_t dim_t;

  // Get inputs
  std::vector<boost::shared_ptr<host_tensor_t> >& inputs = *getInputs();

  // Prepare outputs
  boost::shared_ptr<std::vector<boost::shared_ptr<host_tensor_t> > > outputs(
      new std::vector<boost::shared_ptr<host_tensor_t> >());

  // Load model into device memory
  Model& crbm = *getModel();

  int deviceCount = 0;
  hipGetDeviceCount(&deviceCount);

  const int gpuCount = getGpuCount();

  if (deviceCount < gpuCount) {
    dlog(Severity::Warning) << "Only " << deviceCount << " CUDA-enabled devices found, where " << gpuCount << " are required according to GpuCount. Aborting!";
    return;
  }

  assert(omp_get_num_threads() == 1);

  hipSetDevice(0);
  omp_set_dynamic(0);
  omp_set_num_threads(gpuCount);

  std::vector<boost::shared_ptr<host_tensor_t> >& filters = *crbm.getFilters();
  std::vector<boost::shared_ptr<host_tensor_t> >& c = *crbm.getHiddenBiases();
  tensor_t b = *crbm.getVisibleBias();

  std::vector<boost::shared_ptr<ctensor_t> > cF(filters.size()), cc(filters.size());
  tensor_t output, v_master;
  ctensor_t cv_master;

  #pragma omp parallel
  {
    plan_t plan_v, iplan_v, plan_h, iplan_h;

    /*** PREPARE GPU THREADS ***/

    int tid = omp_get_thread_num();
    hipSetDevice(tid);

    // Enable peer to peer access of each card with the master card and vice versa
    if (tid == 0) {
      for (int i = 1; i < gpuCount; ++i)
        hipDeviceEnablePeerAccess(i, 0);
    } else {
      hipDeviceEnablePeerAccess(0, 0);
    }
    #pragma omp barrier

    // Copy filters to the device and pre-calculate the FFT
    {
      tensor_t f, h;
      ctensor_t cf, ch;
      for (size_t k = tid; k < filters.size(); k += gpuCount) {
        f = *filters[k];
        cf = fft(f, dimCount - 1, plan_v);
        cF[k] = boost::make_shared<ctensor_t>(cf);

        h = *c[k];
        ch = fft(h, dimCount - 1, plan_h);
        cc[k] = boost::make_shared<ctensor_t>(ch);
      }
    }

    tensor_t v, h;
    ctensor_t cv, ch_full, ch;

    for (size_t i = 0; i < inputs.size() && (monitor ? !monitor->getAbortRequested() : true); ++i) {

      if (getDirection() == CodingDirection::Encode) {

        hipStreamSynchronize(0);
        #pragma omp barrier

        #pragma omp master
        {
          v_master = *inputs[i];
          if (crbm.getVisibleUnitType() == UnitType::Gaussian)
            v_master = (v_master - crbm.getMean()) / crbm.getStddev();
          cv_master = fft(v_master, dimCount - 1, plan_v);
          output.resize(seq(v_master.size()[0], v_master.size()[1], v_master.size()[2], (int)cF.size()),
              seq(v_master.size()[0], v_master.size()[1], v_master.size()[2], (int)cF.size()));
          hipStreamSynchronize(0);
        }
        #pragma omp barrier

        cv = cv_master;

        bool validSize = true;
        for (unsigned j = 0; j < dimCount - 1; ++j) {
          if (v_master.size()[j] != upper_power_of_two(v_master.size()[j])) {
            dlog(Severity::Warning) << "The input size in each dimension must be a power of 2. Skipping image!";
            validSize = false;
            break;
          }
        }
        if (!validSize)
          continue;

        for (size_t k = tid; k < cF.size(); k += gpuCount) {
          ch_full = conj(*cF[k]) * cv;
          ch = sum(ch_full, dimCount - 1);
          ch = ch + *cc[k];
          h = ifft(ch, dimCount - 1, iplan_h);

          switch (crbm.getHiddenUnitType()) {
            case UnitType::Bernoulli: h = sigm(h); break;
            case UnitType::ReLU:      h = max(0.0, h);  break;
            case UnitType::MyReLU:    h = nrelu_mean(h); break;
            case UnitType::ReLU1:     h = min(1.0, max(0.0, h));  break;
            case UnitType::ReLU2:     h = min(2.0, max(0.0, h));  break;
            case UnitType::ReLU4:     h = min(4.0, max(0.0, h));  break;
            case UnitType::ReLU8:     h = min(8.0, max(0.0, h));  break;
            default:
              dlog(Severity::Warning) << "Unsupported hidden unit type: " << crbm.getVisibleUnitType();
          }
          output[seq(0,0,0,(int)k), h.size()] = h;
        }
        hipStreamSynchronize(0);
        #pragma omp barrier

      } else {  /* getDirection() == Decoding */

        cv = zeros<complex_t>(cF[0]->size(), cF[0]->fullsize());

        #pragma omp master
        {
          cv_master = zeros<complex_t>(cF[0]->size(), cF[0]->fullsize());
          hipStreamSynchronize(0);
        }
        #pragma omp barrier

        for (size_t k = tid; k < cF.size(); k += gpuCount) {
          h = (*inputs[i])[seq(0,0,0,(int)k), seq(inputs[i]->size()[0],inputs[i]->size()[1], inputs[i]->size()[2],1)];
          ch = fft(h, dimCount - 1, plan_h);

          cv = cv + *cF[k] * repeat(ch, cF[k]->size() / ch.size());
        }

        #pragma omp critical
        {
          cv_master = cv_master + cv;
          hipStreamSynchronize(0);
        }
        #pragma omp barrier

        #pragma omp master
        {
          v = ifft(cv_master, dimCount - 1, iplan_v);

          switch(crbm.getVisibleUnitType()) {
            case UnitType::Bernoulli: v = sigm(v + b); break;
            case UnitType::Gaussian:  v = v + b;       break;
            default:
              dlog(Severity::Warning) << "Unsupported unit type: " << crbm.getVisibleUnitType();
          }
          output = (v * crbm.getStddev()) + crbm.getMean();
          hipStreamSynchronize(0);
        }
        #pragma omp barrier
      }

      #pragma omp master
      {
        outputs->push_back(boost::make_shared<host_tensor_t>(output));
        if (monitor)
          monitor->reportProgress(100. * i / inputs.size());
      }
    }

    hipStreamSynchronize(0);
    #pragma omp barrier

    // Free up memory
    for (size_t k = tid; k < cF.size(); k += gpuCount) {
      cF[k] = cc[k] = boost::shared_ptr<ctensor_t>();
    }

    if (tid == 0) {
      for (int i = 1; i < gpuCount; ++i)
        hipDeviceDisablePeerAccess(i);
    } else {
      hipDeviceDisablePeerAccess(0);
    }
  } /* end of parallel */

  newState->setOutputs(outputs);
}

}

}
