#include "hip/hip_runtime.h"
/*
 * Initialize_gpu.cu
 *
 *  Created on: Nov 22, 2012
 *      Author: tombr
 */

#include "Initialize.h"

#include <tbblas/sum.hpp>
#include <tbblas/dot.hpp>
#include <tbblas/random.hpp>
#include <tbblas/gaussian.hpp>
#include <tbblas/zeros.hpp>
#include <tbblas/ones.hpp>
#include <tbblas/repeat.hpp>

namespace gml {

namespace convrbm4d {

InitializeChecker::InitializeChecker() {
  Initialize test;
  test.initializeClass();
  CHECK_MEMORY_LAYOUT2(Tensors, test);
  CHECK_MEMORY_LAYOUT2(Mask, test);
  CHECK_MEMORY_LAYOUT2(FilterWidth, test);
  CHECK_MEMORY_LAYOUT2(FilterHeight, test);
  CHECK_MEMORY_LAYOUT2(FilterDepth, test);
  CHECK_MEMORY_LAYOUT2(FilterCount, test);
  CHECK_MEMORY_LAYOUT2(WeightMean, test);
  CHECK_MEMORY_LAYOUT2(WeightStddev, test);
  CHECK_MEMORY_LAYOUT2(VisibleUnitType, test);
  CHECK_MEMORY_LAYOUT2(HiddenUnitType, test);
  CHECK_MEMORY_LAYOUT2(ConvolutionType, test);

  CHECK_MEMORY_LAYOUT2(Model, test);
}

void Initialize::update(gapputils::workflow::IProgressMonitor* monitor) const {
  using namespace tbblas;
  using namespace tbblas::deeplearn;

  typedef host_tensor_t::value_t value_t;

  Logbook& dlog = getLogbook();

  // Calculate the mean and the std of all features
  const int filterCount = getFilterCount();
  const int dimCount = host_tensor_t::dimCount;

  boost::shared_ptr<model_t> crbm(new model_t());
  crbm->set_visibles_type(getVisibleUnitType());
  crbm->set_hiddens_type(getHiddenUnitType());
  crbm->set_convolution_type(getConvolutionType());

  v_host_tensor_t& tensors = *getTensors();
  host_tensor_t::dim_t size = tensors[0]->size(), maskSize = size;
  maskSize[dimCount - 1] = 1;

  host_tensor_t mask = (getMask() ? *getMask() : ones<value_t>(maskSize));
  const value_t count = sum(mask) * size[dimCount - 1];

  if (!(mask.size() == maskSize)) {
    dlog(Severity::Warning) << "Size mismatch between input tensors and mask. Aborting!";
    return;
  }

  const int totalCount = tensors.size() * 2 + filterCount;

  if (getVisibleUnitType() == unit_type::Gaussian) {

    // Calculate the mean and normalize the data
    value_t mean = 0;
    for (size_t i = 0; i < tensors.size(); ++i) {
      mean = mean + sum(*tensors[i] * repeat(mask, size / maskSize)) / count;
      if (monitor)
        monitor->reportProgress(100.0 * i / totalCount);
    }
    mean /= tensors.size();

    // Calculate the stddev and normalize the data
    value_t var = 0;
    for (size_t i = 0; i < tensors.size(); ++i) {
      var += dot((*tensors[i] - mean) * repeat(mask, size / maskSize), (*tensors[i] - mean) * repeat(mask, size / maskSize)) / count;
      if (monitor)
        monitor->reportProgress(100.0 * (i + tensors.size()) / totalCount);
    }

    value_t stddev = sqrt(var / tensors.size());
    crbm->set_mean(mean);
    crbm->set_stddev(stddev);
  } else {
    crbm->set_mean(0.0);
    crbm->set_stddev(1.0);
  }

  // Initialize filters and bias terms
  host_tensor_t vb = zeros<value_t>(tensors[0]->size());
  v_host_tensor_t hb;
  v_host_tensor_t filters;

  host_tensor_t::dim_t kernelSize;
  kernelSize[0] = getFilterWidth();
  kernelSize[1] = getFilterHeight();
  kernelSize[2] = getFilterDepth();
  kernelSize[3] = size[3];

  random_tensor<value_t, model_t::dimCount, false, normal<value_t> > randn(kernelSize);
  host_tensor_t sample;

  host_tensor_t::dim_t hiddenSize = tensors[0]->size();
  hiddenSize[model_t::dimCount - 1] = 1;

  for (int i = 0; i < filterCount; ++i) {
    sample = (getWeightStddev() * randn + getWeightMean()); // / (value_t)randn.count();
    filters.push_back(boost::make_shared<host_tensor_t>(sample));
    hb.push_back(boost::make_shared<host_tensor_t>(zeros<value_t>(hiddenSize)));
    if (monitor)
      monitor->reportProgress(100.0 * (i + 2 * tensors.size()) / totalCount);
  }

  crbm->set_filters(filters);
  crbm->set_hidden_bias(hb);
  crbm->set_visible_bias(vb);
  crbm->set_kernel_size(kernelSize);
  crbm->set_mask(mask);

  newState->setModel(crbm);
}

}

}
