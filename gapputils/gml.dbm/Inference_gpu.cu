#include "hip/hip_runtime.h"
/*
 * Inference_gpu.cu
 *
 *  Created on: Jul 12, 2013
 *      Author: tombr
 */

#include "Inference.h"

#include <tbblas/fft.hpp>
#include <tbblas/math.hpp>
#include <tbblas/zeros.hpp>
#include <tbblas/repeat.hpp>
#include <tbblas/shift.hpp>
#include <tbblas/rearrange.hpp>
#include <tbblas/linalg.hpp>

#include <omp.h>

#include "math.hpp"

namespace gml {

namespace dbm {

InferenceChecker::InferenceChecker() {
  Inference test;
  test.initializeClass();
  CHECK_MEMORY_LAYOUT2(Model, test);
  CHECK_MEMORY_LAYOUT2(Inputs, test);
  CHECK_MEMORY_LAYOUT2(Mode, test);
  CHECK_MEMORY_LAYOUT2(ObservedLayer, test);
  CHECK_MEMORY_LAYOUT2(QueryLayer, test);
  CHECK_MEMORY_LAYOUT2(Iterations, test);
  CHECK_MEMORY_LAYOUT2(GpuCount, test);
  CHECK_MEMORY_LAYOUT2(Outputs, test);
}

void Inference::update(IProgressMonitor* monitor) const {
  // Perform a single up pass to initialize the values
  // Iteratively update

  using namespace tbblas;

  Logbook& dlog = getLogbook();

  const unsigned dimCount = Model::dimCount;
  typedef complex<value_t> complex_t;
  typedef fft_plan<dimCount> plan_t;
  typedef tensor<value_t, dimCount, true> tensor_t;
  typedef tensor<complex_t, dimCount, true> ctensor_t;
  typedef tensor<complex_t, dimCount, false> host_ctensor_t;
  typedef std::vector<boost::shared_ptr<ctensor_t> > v_ctensor_t;
  typedef std::vector<boost::shared_ptr<v_ctensor_t> > vv_ctensor_t;
  typedef tensor_t::dim_t dim_t;

  typedef tensor<value_t, 2, true> matrix_t;

  // Get inputs
  std::vector<boost::shared_ptr<host_tensor_t> >& inputs = *getInputs();

  // Prepare outputs
  boost::shared_ptr<std::vector<boost::shared_ptr<host_tensor_t> > > outputs(
      new std::vector<boost::shared_ptr<host_tensor_t> >());

  // Load model into device memory
  Model& dbm = *getModel();

  // A DBM with 1 visible layer and n hidden layers has n layers for the sake of writing this code
  size_t cLayerCount = dbm.getWeights()->size();
  size_t rLayerCount = dbm.getWeightMatrices()->size();
  assert(cLayerCount && rLayerCount);

  dim_t visSize[cLayerCount], hidSize[cLayerCount], layerSize[cLayerCount];
  for (size_t iLayer = 0; iLayer < cLayerCount; ++iLayer) {
    visSize[iLayer] = hidSize[iLayer] = layerSize[iLayer] = dbm.getHiddenBiases()->at(iLayer)->at(0)->size();
    visSize[iLayer][dimCount - 1] = dbm.getWeights()->at(iLayer)->at(0)->size()[dimCount - 1];
    hidSize[iLayer][dimCount - 1] = dbm.getWeights()->at(iLayer)->size();
  }

  dim_t rearrangeBlock[cLayerCount];
  rearrangeBlock[0] = dbm.getVisibleBlockSize();
  for (size_t iLayer = 1; iLayer < cLayerCount; ++iLayer)
    rearrangeBlock[iLayer] = dbm.getHiddenBiases()->at(iLayer - 1)->at(0)->size() / dbm.getHiddenBiases()->at(iLayer)->at(0)->size();

  int deviceCount = 0;
  hipGetDeviceCount(&deviceCount);

  const int gpuCount = getGpuCount();

  // Setting defaults

  int observedLayer, queryLayer;

  switch (getMode()) {
  case InferenceMode::BottomUp:
    observedLayer = (getObservedLayer() < 0 ? 0 : getObservedLayer());
    queryLayer = (getQueryLayer() < 0 ? cLayerCount + rLayerCount : getQueryLayer());
    break;

  case InferenceMode::TopDown:
    observedLayer = (getObservedLayer() < 0 ? cLayerCount + rLayerCount : getObservedLayer());
    queryLayer = (getQueryLayer() < 0 ? 0 : getQueryLayer());
    break;
  }

  // Perform sanity checks

  if (deviceCount < gpuCount) {
    dlog(Severity::Warning) << "Only " << deviceCount << " CUDA-enabled devices found, where " << gpuCount << " are required according to GpuCount. Aborting!";
    return;
  }

  switch (getMode()) {
  case InferenceMode::BottomUp:
    if (observedLayer >= queryLayer) {
      dlog(Severity::Warning) << "The observed layer must be below the query layer for bottom-up inference. Aborting!";
      return;
    }
    break;

  case InferenceMode::TopDown:
    if (observedLayer <= queryLayer) {
      dlog(Severity::Warning) << "The observed layer must be above the query layer for top-down inference. Aborting!";
      return;
    }
    break;
  }

  assert(omp_get_num_threads() == 1);

  hipSetDevice(0);
  omp_set_dynamic(0);
  omp_set_num_threads(gpuCount);

  vv_host_tensor_t& filters = *dbm.getWeights();
  vv_host_tensor_t& c = *dbm.getHiddenBiases();
  tensor_t b = *dbm.getVisibleBias();

  v_ctensor_t cF[cLayerCount], cc[cLayerCount];
  for (size_t iLayer = 0; iLayer < cLayerCount; ++iLayer) {
    cF[iLayer].resize(filters[iLayer]->size());
    cc[iLayer].resize(filters[iLayer]->size());
  }

  matrix_t W[rLayerCount], c_flat[rLayerCount];
  for (size_t iLayer = 0; iLayer < rLayerCount; ++iLayer) {
    W[iLayer] = *dbm.getWeightMatrices()->at(iLayer);
    c_flat[iLayer] = *dbm.getFlatBiases()->at(iLayer);
  }

  tensor_t v_master[cLayerCount + 1], V_master[cLayerCount];
  ctensor_t cV_master[cLayerCount];
  matrix_t v_flat[rLayerCount + 1], h_flat[rLayerCount];

  #pragma omp parallel
  {
    /*** PREPARE GPU THREADS ***/

    int tid = omp_get_thread_num();
    hipSetDevice(tid);

    // Enable peer to peer access of each card with the master card and vice versa
    if (tid == 0) {
      for (int i = 1; i < gpuCount; ++i)
        hipDeviceEnablePeerAccess(i, 0);
    } else {
      hipDeviceEnablePeerAccess(0, 0);
    }
    #pragma omp barrier

    plan_t plan_v[cLayerCount], iplan_v[cLayerCount], plan_h[cLayerCount], iplan_h[cLayerCount];
    tensor_t hMask[cLayerCount];

    for (size_t iLayer = 0; iLayer < cLayerCount; ++iLayer) {
      hMask[iLayer] = *dbm.getMasks()->at(iLayer);

      // Copy filters to the device and pre-calculate the FFT
      {
        tensor_t f, h, kern, pad;
        ctensor_t cf, ch;
        for (size_t k = tid; k < filters[iLayer]->size(); k += gpuCount) {

          kern = *filters[iLayer]->at(k);
          dim_t topleft = visSize[iLayer] / 2 - kern.size() / 2;
          pad = zeros<value_t>(visSize[iLayer]);
          pad[topleft, kern.size()] = kern;
          f = ifftshift(pad, dimCount - 1);
          cf = fft(f, dimCount - 1, plan_v[iLayer]);
          cF[iLayer][k] = boost::make_shared<ctensor_t>(cf);

          h = *c[iLayer]->at(k);
          ch = fft(h, dimCount - 1, plan_h[iLayer]);
          cc[iLayer][k] = boost::make_shared<ctensor_t>(ch);
        }
      }
    }

    tensor_t h[cLayerCount];
    ctensor_t cV[cLayerCount], ch_full[cLayerCount], ch[cLayerCount];

    for (size_t i = 0; i < inputs.size() && (monitor ? !monitor->getAbortRequested() : true); ++i) {

      if (getMode() == InferenceMode::BottomUp) {

        /*** LOAD VISIBLE LAYER ***/

        hipStreamSynchronize(0);
        #pragma omp barrier

        #pragma omp master
        {
          // Initialize the flat layer first because this also reserves space in case the flat layer is observed
          for (size_t iLayer = max(0, observedLayer - (int)cLayerCount); iLayer < rLayerCount + 1; ++iLayer) {
            if (iLayer == 0)
              v_flat[iLayer] = zeros<value_t>(seq(1, (int)hidSize[cLayerCount - 1].count()));
            else
              v_flat[iLayer] = zeros<value_t>(c_flat[iLayer - 1].size());
          }

          if (observedLayer < cLayerCount) {
            v_master[observedLayer] = *inputs[i];
            V_master[observedLayer] = rearrange(v_master[observedLayer], rearrangeBlock[observedLayer]);
            if (observedLayer == 0)
              V_master[observedLayer] = (V_master[observedLayer] - dbm.getMean()) / dbm.getStddev();
            V_master[observedLayer] = V_master[observedLayer] * repeat(hMask[observedLayer], visSize[observedLayer] / layerSize[observedLayer]);
            cV_master[observedLayer] = fft(V_master[observedLayer], dimCount - 1, plan_v[observedLayer]);

            for (size_t iLayer = observedLayer; iLayer < cLayerCount; ++iLayer)
              v_master[iLayer + 1] = zeros<value_t>(hidSize[iLayer]);
          } else {
            thrust::copy(inputs[i]->begin(), inputs[i]->end(), v_flat[observedLayer - cLayerCount].begin());
          }

          hipStreamSynchronize(0);
        }
        #pragma omp barrier

        if (observedLayer < cLayerCount)
          cV[observedLayer] = cV_master[observedLayer];

        // Perform multiple mean field updates (first update initializes the model)
        for (size_t iMeanField = 0; iMeanField < getIterations(); ++iMeanField) {

          // Go through convolutional layers first
          for (size_t iLayer = observedLayer; iLayer < cLayerCount; ++iLayer) {

            // If not the top-most layer, calculate top-down signal from convolutional layer
            if (iLayer < cLayerCount - 1) {

              cV[iLayer + 1] = zeros<complex_t>(cF[iLayer + 1][0]->size(), cF[iLayer + 1][0]->fullsize());

              #pragma omp master
              {
                cV_master[iLayer + 1] = zeros<complex_t>(cF[iLayer + 1][0]->size(), cF[iLayer + 1][0]->fullsize());
                hipStreamSynchronize(0);
              }
              #pragma omp barrier

              for (size_t k = tid; k < cF[iLayer + 1].size(); k += gpuCount) {
                h[iLayer + 1] = v_master[iLayer + 2][seq(0,0,0,(int)k), layerSize[iLayer + 1]];
                ch[iLayer + 1] = fft(h[iLayer + 1], dimCount - 1, plan_h[iLayer + 1]);

                cV[iLayer + 1] = cV[iLayer + 1] + *cF[iLayer + 1][k] * repeat(ch[iLayer + 1], cF[iLayer + 1][k]->size() / ch[iLayer + 1].size());
              }

              #pragma omp critical
              {
                cV_master[iLayer + 1] = cV_master[iLayer + 1] + cV[iLayer + 1];
                hipStreamSynchronize(0);
              }
              #pragma omp barrier

              #pragma omp master
              {
                V_master[iLayer + 1] = ifft(cV_master[iLayer + 1], dimCount - 1, iplan_v[iLayer + 1]);
                v_master[iLayer + 1] = rearrange_r(V_master[iLayer + 1], rearrangeBlock[iLayer + 1]);
                hipStreamSynchronize(0);
              }
            } else {  // calculate top-down signal from first RBM
              #pragma omp master
              {
                v_flat[0] = prod(v_flat[1], tbblas::trans(W[0]));
                assert(v_flat[0].count() == v_master[iLayer + 1].count());
                thrust::copy(v_flat[0].begin(), v_flat[0].end(), v_master[iLayer + 1].begin());
                hipStreamSynchronize(0);
              }
            }
            #pragma omp barrier

            // bottom-up signal
            for (size_t k = tid; k < cF[iLayer].size(); k += gpuCount) {
              if (iMeanField == 0)  // double weights because I'm getting zero input from the upper layer
                ch_full[iLayer] = conj(*cF[iLayer][k]) * cV[iLayer] * 2.0;
              else
                ch_full[iLayer] = conj(*cF[iLayer][k]) * cV[iLayer];
              ch[iLayer] = sum(ch_full[iLayer], dimCount - 1);
              ch[iLayer] = ch[iLayer] + *cc[iLayer][k];
              h[iLayer] = ifft(ch[iLayer], dimCount - 1, iplan_h[iLayer]);
              h[iLayer] = h[iLayer] + v_master[iLayer + 1][seq(0,0,0,(int)k), layerSize[iLayer]];
              h[iLayer] = nrelu_mean(h[iLayer]);
              v_master[iLayer + 1][seq(0,0,0,(int)k), layerSize[iLayer]] = h[iLayer] * hMask[iLayer];
            }
            hipStreamSynchronize(0);
            #pragma omp barrier

            if (iLayer < cLayerCount - 1) {
              // rearrange into master first and then let all threads read from master into cV
              #pragma omp master
              {
                V_master[iLayer + 1] = rearrange(v_master[iLayer + 1], rearrangeBlock[iLayer + 1]);
                cV_master[iLayer + 1] = fft(V_master[iLayer + 1], dimCount - 1, plan_v[iLayer + 1]);
                hipStreamSynchronize(0);
              }
              #pragma omp barrier
              cV[iLayer + 1] = cV_master[iLayer + 1];
            } else {
              #pragma omp master
              thrust::copy(v_master[iLayer + 1].begin(), v_master[iLayer + 1].end(), v_flat[0].begin());
            }
            hipStreamSynchronize(0);
            #pragma omp barrier
          }

          // Then go through RBM layer
          #pragma omp master
          {
            for (size_t iLayer = std::max(0, observedLayer - (int)cLayerCount); iLayer < rLayerCount; ++iLayer) {

              // bottom-up signal
              h_flat[iLayer] = prod(v_flat[iLayer], W[iLayer]);

              if (iLayer < rLayerCount - 1) {  // add top-down signal and bias
                v_flat[iLayer + 1] = prod(v_flat[iLayer + 2], tbblas::trans(W[iLayer + 1]));
                if (iMeanField == 0)
                  v_flat[iLayer + 1] = nrelu_mean(v_flat[iLayer + 1] + 2.0 * h_flat[iLayer] + c_flat[iLayer]);
                else
                  v_flat[iLayer + 1] = nrelu_mean(v_flat[iLayer + 1] + h_flat[iLayer] + c_flat[iLayer]);
              } else {                         // add bias only
                v_flat[iLayer + 1] = nrelu_mean(h_flat[iLayer] + c_flat[iLayer]);
              }
            }
            hipStreamSynchronize(0);
          }
          #pragma omp barrier
        }

        #pragma omp master
        {
          if (queryLayer <= cLayerCount) {
            outputs->push_back(boost::make_shared<host_tensor_t>(v_master[queryLayer]));
          } else {
            boost::shared_ptr<host_tensor_t> output(new host_tensor_t(1, 1, 1, v_flat[queryLayer - cLayerCount].count()));
            thrust::copy(v_flat[queryLayer - cLayerCount].begin(), v_flat[queryLayer - cLayerCount].end(), output->begin());
            outputs->push_back(output);
          }
        }
      } else if (getMode() == InferenceMode::TopDown) { /*** Top-down case ***/

        /*** LOAD HIDDEN LAYER ***/

        for (size_t iLayer = 0; iLayer < min(observedLayer - 1, (int)cLayerCount); ++iLayer)
          cV[iLayer] = zeros<complex_t>(cF[iLayer][0]->size(), cF[iLayer][0]->fullsize());

        #pragma omp master
        {

          for (int iLayer = 0; iLayer < observedLayer - (int)cLayerCount + 1; ++iLayer) {
            if (iLayer == 0)
              v_flat[iLayer] = zeros<value_t>(seq(1, (int)hidSize[cLayerCount - 1].count()));
            else
              v_flat[iLayer] = zeros<value_t>(c_flat[iLayer - 1].size());
          }

          if (observedLayer > cLayerCount) {
            thrust::copy(inputs[i]->begin(), inputs[i]->end(), v_flat[observedLayer - cLayerCount].begin());
          } else {
            v_master[observedLayer] = *inputs[i];
          }

          if (observedLayer > cLayerCount)
            v_master[cLayerCount] = zeros<value_t>(hidSize[cLayerCount - 1]);

          for (size_t iLayer = 0; iLayer < min(observedLayer - 1, (int)cLayerCount); ++iLayer) {
            cV_master[iLayer] = zeros<complex_t>(cF[iLayer][0]->size(), cF[iLayer][0]->fullsize());
          }
          hipStreamSynchronize(0);
        }
        #pragma omp barrier

        // Perform multiple mean field updates (first update initialize the model)
        for (size_t iMeanField = 0; iMeanField < getIterations(); ++iMeanField) {

          // Update RBM layers first
          // this will also update v_master[cLayerCount]
          #pragma omp master
          {
            for (int iLayer = observedLayer - cLayerCount - 2; iLayer >= 0; --iLayer) {

              // bottom-up signal
              h_flat[iLayer] = prod(v_flat[iLayer], W[iLayer]);

              v_flat[iLayer + 1] = prod(v_flat[iLayer + 2], tbblas::trans(W[iLayer + 1]));
              if (iMeanField == 0)
                v_flat[iLayer + 1] = nrelu_mean(2.0 * v_flat[iLayer + 1] + h_flat[iLayer] + c_flat[iLayer]);
              else
                v_flat[iLayer + 1] = nrelu_mean(v_flat[iLayer + 1] + h_flat[iLayer] + c_flat[iLayer]);
            }
            hipStreamSynchronize(0);
          }
          #pragma omp barrier

          for (int iLayer = min(observedLayer - 1, (int)cLayerCount); iLayer >= 0; --iLayer) {

            // If not the top-most layer, calculate convolutional top-down signal
            if (iLayer < cLayerCount) {
              cV[iLayer] = zeros<complex_t>(cF[iLayer][0]->size(), cF[iLayer][0]->fullsize());

              #pragma omp master
              {
                cV_master[iLayer] = zeros<complex_t>(cF[iLayer][0]->size(), cF[iLayer][0]->fullsize());
                hipStreamSynchronize(0);
              }
              #pragma omp barrier

              for (size_t k = tid; k < cF[iLayer].size(); k += gpuCount) {
                h[iLayer] = v_master[iLayer + 1][seq(0,0,0,(int)k), layerSize[iLayer]];
                ch[iLayer] = fft(h[iLayer], dimCount - 1, plan_h[iLayer]);

                if (iMeanField == 0 && iLayer > 0)
                  cV[iLayer] = cV[iLayer] + *cF[iLayer][k] * repeat(ch[iLayer], cF[iLayer][k]->size() / ch[iLayer].size()) * 2.0;
                else
                  cV[iLayer] = cV[iLayer] + *cF[iLayer][k] * repeat(ch[iLayer], cF[iLayer][k]->size() / ch[iLayer].size());
              }

              #pragma omp critical
              {
                cV_master[iLayer] = cV_master[iLayer] + cV[iLayer];
                hipStreamSynchronize(0);
              }
              #pragma omp barrier

              #pragma omp master
              {
                V_master[iLayer] = ifft(cV_master[iLayer], dimCount - 1, iplan_v[iLayer]);
                if (iLayer == 0)
                  V_master[0] = (V_master[0] + b) * repeat(hMask[0], visSize[0] / layerSize[0]);

                v_master[iLayer] = rearrange_r(V_master[iLayer], rearrangeBlock[iLayer]);
                hipStreamSynchronize(0);
              }
            } else {  // calculate top-down signal from first RBM
              #pragma omp master
              {
                v_flat[0] = prod(v_flat[1], tbblas::trans(W[0]));
                assert(v_flat[0].count() == v_master[iLayer].count());
                if (iMeanField == 0)
                  v_flat[0] = 2.0 * v_flat[0];
                thrust::copy(v_flat[0].begin(), v_flat[0].end(), v_master[iLayer].begin());
                hipStreamSynchronize(0);
              }
            }
            #pragma omp barrier

            // bottom-up signal
            if (iLayer > 0) {
              for (size_t k = tid; k < cF[iLayer - 1].size(); k += gpuCount) {
                ch_full[iLayer - 1] = conj(*cF[iLayer - 1][k]) * cV[iLayer - 1];
                ch[iLayer - 1] = sum(ch_full[iLayer - 1], dimCount - 1);
                ch[iLayer - 1] = ch[iLayer - 1] + *cc[iLayer - 1][k];
                h[iLayer - 1] = ifft(ch[iLayer - 1], dimCount - 1, iplan_h[iLayer - 1]);
                h[iLayer - 1] = h[iLayer - 1] + v_master[iLayer][seq(0,0,0,(int)k), layerSize[iLayer - 1]];
                h[iLayer - 1] = nrelu_mean(h[iLayer - 1]);
                v_master[iLayer][seq(0,0,0,(int)k), layerSize[iLayer - 1]] = h[iLayer - 1] * hMask[iLayer - 1];
              }
              hipStreamSynchronize(0);
              #pragma omp barrier
            }

            if (iLayer < cLayerCount) {
              // rearrange into master first and then let all threads read from master into cV
              #pragma omp master
              {
                V_master[iLayer] = rearrange(v_master[iLayer], rearrangeBlock[iLayer]);
                cV_master[iLayer] = fft(V_master[iLayer], dimCount - 1, plan_v[iLayer]);
                hipStreamSynchronize(0);
              }
              #pragma omp barrier
              cV[iLayer] = cV_master[iLayer];

            } else {
              #pragma omp master
              thrust::copy(v_master[iLayer].begin(), v_master[iLayer].end(), v_flat[0].begin());
            }
            hipStreamSynchronize(0);
            #pragma omp barrier
          }
        }

        #pragma omp master
        {
          if (queryLayer <= cLayerCount) {
            if (queryLayer == 0) {
              V_master[0] = (V_master[0] * dbm.getStddev() + dbm.getMean()) * repeat(hMask[0], visSize[0] / layerSize[0]);
              v_master[0] = rearrange_r(V_master[0], rearrangeBlock[0]);
            }
            outputs->push_back(boost::make_shared<host_tensor_t>(v_master[queryLayer]));
          } else {
            boost::shared_ptr<host_tensor_t> output(new host_tensor_t(1, 1, 1, v_flat[queryLayer - cLayerCount].count()));
            thrust::copy(v_flat[queryLayer - cLayerCount].begin(), v_flat[queryLayer - cLayerCount].end(), output->begin());
            outputs->push_back(output);
          }
        }
      }

      #pragma omp master
      if (monitor)
        monitor->reportProgress(100. * (i + 1) / inputs.size());
    }

    hipStreamSynchronize(0);
    #pragma omp barrier

    // Free up memory
    for (size_t iLayer = 0; iLayer < cLayerCount; ++iLayer) {
      for (size_t k = tid; k < cF[iLayer].size(); k += gpuCount) {
        cF[iLayer][k] = cc[iLayer][k] = boost::shared_ptr<ctensor_t>();
      }
    }

    if (tid == 0) {
      for (int i = 1; i < gpuCount; ++i)
        hipDeviceDisablePeerAccess(i);
    } else {
      hipDeviceDisablePeerAccess(0);
    }

  } /* end of parallel */

  newState->setOutputs(outputs);
}

}

}

