#include "hip/hip_runtime.h"
/*
 * Sampler_gpu.cu
 *
 *  Created on: Jul 15, 2013
 *      Author: tombr
 */

#include "Sampler.h"

#include <tbblas/fft.hpp>
#include <tbblas/math.hpp>
#include <tbblas/zeros.hpp>
#include <tbblas/repeat.hpp>
#include <tbblas/shift.hpp>
#include <tbblas/rearrange.hpp>
#include <tbblas/random.hpp>
#include <tbblas/linalg.hpp>

#include <omp.h>

#include "math.hpp"

namespace gml {

namespace dbm {

SamplerChecker::SamplerChecker() {
  Sampler test;
  test.initializeClass();
  CHECK_MEMORY_LAYOUT2(Model, test);
  CHECK_MEMORY_LAYOUT2(GpuCount, test);
  CHECK_MEMORY_LAYOUT2(SampleCount, test);
  CHECK_MEMORY_LAYOUT2(Iterations, test);
  CHECK_MEMORY_LAYOUT2(Damped, test);
  CHECK_MEMORY_LAYOUT2(Samples, test);
}

void Sampler::update(IProgressMonitor* monitor) const {
  // Initialize the visible units with random values from a unit Gaussian
  // This makes sense because the visible units are Gauss distributed with unit variance
  // Perform a single up pass to initialize the values (sampling)
  // Iteratively perform Gibbs sampling in a bottom-up-down manner

  using namespace tbblas;

  Logbook& dlog = getLogbook();

  const unsigned dimCount = Model::dimCount;
  typedef Model::value_t value_t;
  typedef complex<value_t> complex_t;
  typedef fft_plan<dimCount> plan_t;
  typedef tensor<value_t, dimCount, true> tensor_t;
  typedef tensor<complex_t, dimCount, true> ctensor_t;
  typedef tensor<complex_t, dimCount, false> host_ctensor_t;
  typedef std::vector<boost::shared_ptr<ctensor_t> > v_ctensor_t;
  typedef std::vector<boost::shared_ptr<v_ctensor_t> > vv_ctensor_t;
  typedef tensor_t::dim_t dim_t;

  typedef tensor<value_t, 2, true> matrix_t;

  // Prepare outputs
  boost::shared_ptr<v_host_tensor_t> outputs(new v_host_tensor_t());

  // Load model into device memory
  Model& dbm = *getModel();

  // A DBM with 1 visible layer and n hidden layers has n layers for the sake of writing this code
  size_t cLayerCount = dbm.getWeights()->size();
  size_t rLayerCount = dbm.getWeightMatrices()->size();
//  assert(cLayerCount && rLayerCount);
  assert(cLayerCount);

  dim_t visSize[cLayerCount], hidSize[cLayerCount], layerSize[cLayerCount];
  for (size_t iLayer = 0; iLayer < cLayerCount; ++iLayer) {
    visSize[iLayer] = hidSize[iLayer] = layerSize[iLayer] = dbm.getHiddenBiases()->at(iLayer)->at(0)->size();
    visSize[iLayer][dimCount - 1] = dbm.getWeights()->at(iLayer)->at(0)->size()[dimCount - 1];
    hidSize[iLayer][dimCount - 1] = dbm.getWeights()->at(iLayer)->size();
  }

  dim_t rearrangeBlock[cLayerCount];
  rearrangeBlock[0] = dbm.getVisibleBlockSize();
  for (size_t iLayer = 1; iLayer < cLayerCount; ++iLayer)
    rearrangeBlock[iLayer] = dbm.getHiddenBiases()->at(iLayer - 1)->at(0)->size() / dbm.getHiddenBiases()->at(iLayer)->at(0)->size();

  int deviceCount = 0;
  hipGetDeviceCount(&deviceCount);

  const int gpuCount = getGpuCount();

  if (deviceCount < gpuCount) {
    dlog(Severity::Warning) << "Only " << deviceCount << " CUDA-enabled devices found, where " << gpuCount << " are required according to GpuCount. Aborting!";
    return;
  }

  assert(omp_get_num_threads() == 1);

  hipSetDevice(0);
  omp_set_dynamic(0);
  omp_set_num_threads(gpuCount);

  vv_host_tensor_t& filters = *dbm.getWeights();
  vv_host_tensor_t& c = *dbm.getHiddenBiases();
  tensor_t b = *dbm.getVisibleBias();

  v_ctensor_t cF[cLayerCount], cc[cLayerCount];
  for (size_t iLayer = 0; iLayer < cLayerCount; ++iLayer) {
    cF[iLayer].resize(filters[iLayer]->size());
    cc[iLayer].resize(filters[iLayer]->size());
  }

  matrix_t W[rLayerCount], c_flat[rLayerCount];
  for (size_t iLayer = 0; iLayer < rLayerCount; ++iLayer) {
    W[iLayer] = *dbm.getWeightMatrices()->at(iLayer);
    c_flat[iLayer] = *dbm.getFlatBiases()->at(iLayer);
  }

  random_tensor<value_t, 2, true, normal<value_t> > flat_noise[rLayerCount];
  for (size_t iLayer = 0; iLayer < rLayerCount; ++iLayer)
    flat_noise[iLayer].resize(c_flat[iLayer].size());

  tensor_t v_master[cLayerCount + 1], V_master[cLayerCount];
  ctensor_t cV_master[cLayerCount];
  matrix_t v_flat[rLayerCount + 1], h_flat[rLayerCount];

  #pragma omp parallel
  {
    /*** PREPARE GPU THREADS ***/

    int tid = omp_get_thread_num();
    hipSetDevice(tid);

    // Enable peer to peer access of each card with the master card and vice versa
    if (tid == 0) {
      for (int i = 1; i < gpuCount; ++i)
        hipDeviceEnablePeerAccess(i, 0);
    } else {
      hipDeviceEnablePeerAccess(0, 0);
    }
    #pragma omp barrier

    plan_t plan_v[cLayerCount], iplan_v[cLayerCount], plan_h[cLayerCount], iplan_h[cLayerCount];
    tensor_t hMask[cLayerCount];

    random_tensor<value_t, dimCount, true, normal<value_t> > h_noise[cLayerCount];
    for (size_t i = 0; i < cLayerCount; ++i)
      h_noise[i].resize(layerSize[i], tid);
    random_tensor<value_t, dimCount, true, normal<value_t> > V_noise(visSize[0], tid);

    for (size_t iLayer = 0; iLayer < cLayerCount; ++iLayer) {
      hMask[iLayer] = *dbm.getMasks()->at(iLayer);

      // Copy filters to the device and pre-calculate the FFT
      {
        tensor_t f, h, kern, pad;
        ctensor_t cf, ch;
        for (size_t k = tid; k < filters[iLayer]->size(); k += gpuCount) {

          kern = *filters[iLayer]->at(k);
          dim_t topleft = visSize[iLayer] / 2 - kern.size() / 2;
          pad = zeros<value_t>(visSize[iLayer]);
          pad[topleft, kern.size()] = kern;
          f = ifftshift(pad, dimCount - 1);
          cf = fft(f, dimCount - 1, plan_v[iLayer]);
          cF[iLayer][k] = boost::make_shared<ctensor_t>(cf);

          h = *c[iLayer]->at(k);
          ch = fft(h, dimCount - 1, plan_h[iLayer]);
          cc[iLayer][k] = boost::make_shared<ctensor_t>(ch);
        }
      }
    }

    tensor_t h[cLayerCount];
    ctensor_t cV[cLayerCount], ch_full[cLayerCount], ch[cLayerCount];

    for (size_t iSample = 0; iSample < getSampleCount() && (monitor ? !monitor->getAbortRequested() : true); ++iSample) {

      hipStreamSynchronize(0);
      #pragma omp barrier

      #pragma omp master
      {
        V_master[0] = 0.0 * V_noise * repeat(hMask[0], visSize[0] / layerSize[0]);
        cV_master[0] = fft(V_master[0], dimCount - 1, plan_v[0]);

        for (size_t iLayer = 0; iLayer < cLayerCount; ++iLayer)
          v_master[iLayer + 1] = zeros<value_t>(hidSize[iLayer]);

        v_flat[0] = zeros<value_t>(seq(1, (int)v_master[cLayerCount].count()));
        for (size_t iLayer = 0; iLayer < rLayerCount; ++iLayer)
          v_flat[iLayer + 1] = zeros<value_t>(c_flat[iLayer].size());

        hipStreamSynchronize(0);
      }
      #pragma omp barrier

      cV[0] = cV_master[0];

      // Perform multiple Gibbs updates (first update initializes the model)
      for (size_t iGibbs = 0; iGibbs < getIterations(); ++iGibbs) {

        /*** Follow bottom-up Gibbs chain ***/

        for (size_t iLayer = 0; iLayer < cLayerCount; ++iLayer) {

          // If not the top-most layer, calculate convolutional top-down signal
          if (iLayer < cLayerCount - 1) {

            cV[iLayer + 1] = zeros<complex_t>(cF[iLayer + 1][0]->size(), cF[iLayer + 1][0]->fullsize());

            #pragma omp master
            {
              cV_master[iLayer + 1] = zeros<complex_t>(cF[iLayer + 1][0]->size(), cF[iLayer + 1][0]->fullsize());
              hipStreamSynchronize(0);
            }
            #pragma omp barrier

            for (size_t k = tid; k < cF[iLayer + 1].size(); k += gpuCount) {
              h[iLayer + 1] = v_master[iLayer + 2][seq(0,0,0,(int)k), layerSize[iLayer + 1]];
              ch[iLayer + 1] = fft(h[iLayer + 1], dimCount - 1, plan_h[iLayer + 1]);

              cV[iLayer + 1] = cV[iLayer + 1] + *cF[iLayer + 1][k] * repeat(ch[iLayer + 1], cF[iLayer + 1][k]->size() / ch[iLayer + 1].size());
            }

            #pragma omp critical
            {
              cV_master[iLayer + 1] = cV_master[iLayer + 1] + cV[iLayer + 1];
              hipStreamSynchronize(0);
            }
            #pragma omp barrier

            #pragma omp master
            {
              V_master[iLayer + 1] = ifft(cV_master[iLayer + 1], dimCount - 1, iplan_v[iLayer + 1]);
              v_master[iLayer + 1] = rearrange_r(V_master[iLayer + 1], rearrangeBlock[iLayer + 1]);
              hipStreamSynchronize(0);
            }
          } else if (rLayerCount) { // calculate top-down signal from first RBM (if there are RBMs)
            #pragma omp master
            {
              v_flat[0] = prod(v_flat[1], tbblas::trans(W[0]));
              assert(v_flat[0].count() == v_master[iLayer + 1].count());
              thrust::copy(v_flat[0].begin(), v_flat[0].end(), v_master[iLayer + 1].begin());
              hipStreamSynchronize(0);
            }
          }
          #pragma omp barrier

          // bottom-up signal
          for (size_t k = tid; k < cF[iLayer].size(); k += gpuCount) {
            if (iGibbs == 0)  // double weights because I'm getting zero input from the upper layer
              ch_full[iLayer] = conj(*cF[iLayer][k]) * cV[iLayer] * 1.0;
            else
              ch_full[iLayer] = conj(*cF[iLayer][k]) * cV[iLayer];
            ch[iLayer] = sum(ch_full[iLayer], dimCount - 1);
            ch[iLayer] = ch[iLayer] + *cc[iLayer][k];
            h[iLayer] = ifft(ch[iLayer], dimCount - 1, iplan_h[iLayer]);

            if (rLayerCount)
              h[iLayer] = h[iLayer] + v_master[iLayer + 1][seq(0,0,0,(int)k), layerSize[iLayer]];

            h[iLayer] = max(0.0, h[iLayer] + sqrt(sigm(h[iLayer])) * h_noise[iLayer]);
            v_master[iLayer + 1][seq(0,0,0,(int)k), layerSize[iLayer]] = h[iLayer] * hMask[iLayer];
          }
          hipStreamSynchronize(0);
          #pragma omp barrier

          if (iLayer < cLayerCount - 1) {
            // rearrange into master first and then let all threads read from master into cV
            #pragma omp master
            {
              V_master[iLayer + 1] = rearrange(v_master[iLayer + 1], rearrangeBlock[iLayer + 1]);
              cV_master[iLayer + 1] = fft(V_master[iLayer + 1], dimCount - 1, plan_v[iLayer + 1]);
              hipStreamSynchronize(0);
            }
            #pragma omp barrier
            cV[iLayer + 1] = cV_master[iLayer + 1];
          } else if (rLayerCount) {
            #pragma omp master
            thrust::copy(v_master[iLayer + 1].begin(), v_master[iLayer + 1].end(), v_flat[0].begin());
          }
          hipStreamSynchronize(0);
          #pragma omp barrier
        } /* end of bottom-up pass */

        // Then go through RBM layer
        #pragma omp master
        {
          for (size_t iLayer = 0; iLayer < rLayerCount; ++iLayer) {

            // bottom-up signal
            h_flat[iLayer] = prod(v_flat[iLayer], W[iLayer]);

            if (iLayer < rLayerCount - 1) {  // add top-down signal and bias
              v_flat[iLayer + 1] = prod(v_flat[iLayer + 2], tbblas::trans(W[iLayer + 1]));
              if (iGibbs == 0)
                v_flat[iLayer + 1] = v_flat[iLayer + 1] + 1.0 * h_flat[iLayer] + c_flat[iLayer];
              else
                v_flat[iLayer + 1] = v_flat[iLayer + 1] + h_flat[iLayer] + c_flat[iLayer];
            } else {                         // add bias only
              v_flat[iLayer + 1] = h_flat[iLayer] + c_flat[iLayer];
            }
            v_flat[iLayer + 1] = max(0.0, v_flat[iLayer + 1] + sqrt(sigm(v_flat[iLayer + 1])) * flat_noise[iLayer]);
          }
          hipStreamSynchronize(0);
        }
        #pragma omp barrier

        /*** Follow top-down Gibbs chain ***/

        // Update RBM layers first
        #pragma omp master
        {
          for (int iLayer = rLayerCount - 2; iLayer >= 0; --iLayer) {

            // bottom-up signal
            h_flat[iLayer] = prod(v_flat[iLayer], W[iLayer]);

            v_flat[iLayer + 1] = prod(v_flat[iLayer + 2], tbblas::trans(W[iLayer + 1]));
            v_flat[iLayer + 1] = v_flat[iLayer + 1] + h_flat[iLayer] + c_flat[iLayer];

            v_flat[iLayer + 1] = max(0.0, v_flat[iLayer + 1] + sqrt(sigm(v_flat[iLayer + 1])) * flat_noise[iLayer]);
          }
          hipStreamSynchronize(0);
        }
        #pragma omp barrier

        for (int iLayer = cLayerCount; iLayer >= 0; --iLayer) {

          // If not top-most layer, calculate convolutional top-down signal
          if (iLayer < cLayerCount) {
            cV[iLayer] = zeros<complex_t>(cF[iLayer][0]->size(), cF[iLayer][0]->fullsize());

            #pragma omp master
            {
              cV_master[iLayer] = zeros<complex_t>(cF[iLayer][0]->size(), cF[iLayer][0]->fullsize());
              hipStreamSynchronize(0);
            }
            #pragma omp barrier

            for (size_t k = tid; k < cF[iLayer].size(); k += gpuCount) {
              h[iLayer] = v_master[iLayer + 1][seq(0,0,0,(int)k), layerSize[iLayer]];
              ch[iLayer] = fft(h[iLayer], dimCount - 1, plan_h[iLayer]);
              cV[iLayer] = cV[iLayer] + *cF[iLayer][k] * repeat(ch[iLayer], cF[iLayer][k]->size() / ch[iLayer].size());
            }

            #pragma omp critical
            {
              cV_master[iLayer] = cV_master[iLayer] + cV[iLayer];
              hipStreamSynchronize(0);
            }
            #pragma omp barrier

            #pragma omp master
            {
              V_master[iLayer] = ifft(cV_master[iLayer], dimCount - 1, iplan_v[iLayer]);
              if (iLayer == 0) {
                V_master[0] = (V_master[0] + b) * repeat(hMask[0], visSize[0] / layerSize[0]);

                if (getDamped()) {
                  value_t count = sum(hMask[0]) / visSize[0][dimCount - 1];
                  value_t mean = sum(V_master[0]) / count;
                  V_master[0] = (V_master[0] - mean) * repeat(hMask[0], visSize[0] / layerSize[0]);
                  value_t sd = sqrt(sum(V_master[0] * V_master[0]) / count);
                  V_master[0] = V_master[0] / sd;
                }
              }

              v_master[iLayer] = rearrange_r(V_master[iLayer], rearrangeBlock[iLayer]);
              hipStreamSynchronize(0);
            }
          } else if (rLayerCount) { // calculate top-down signal from first RBM
            #pragma omp master
            {
              v_flat[0] = prod(v_flat[1], tbblas::trans(W[0]));
              assert(v_flat[0].count() == v_master[iLayer].count());
              thrust::copy(v_flat[0].begin(), v_flat[0].end(), v_master[iLayer].begin());
              hipStreamSynchronize(0);
            }
          }
          #pragma omp barrier

          // bottom-up signal
          if (iLayer > 0) {
            for (size_t k = tid; k < cF[iLayer - 1].size(); k += gpuCount) {
              ch_full[iLayer - 1] = conj(*cF[iLayer - 1][k]) * cV[iLayer - 1];
              ch[iLayer - 1] = sum(ch_full[iLayer - 1], dimCount - 1);
              ch[iLayer - 1] = ch[iLayer - 1] + *cc[iLayer - 1][k];
              h[iLayer - 1] = ifft(ch[iLayer - 1], dimCount - 1, iplan_h[iLayer - 1]);
              if (rLayerCount)
                h[iLayer - 1] = h[iLayer - 1] + v_master[iLayer][seq(0,0,0,(int)k), layerSize[iLayer - 1]];
              h[iLayer - 1] = max(0.0, h[iLayer - 1] + sqrt(sigm(h[iLayer - 1])) * h_noise[iLayer - 1]);
              v_master[iLayer][seq(0,0,0,(int)k), layerSize[iLayer - 1]] = h[iLayer - 1] * hMask[iLayer - 1];
            }
            hipStreamSynchronize(0);
            #pragma omp barrier
          }

          if (iLayer < cLayerCount) {
            // rearrange into master first and then let all threads read from master into cV
            #pragma omp master
            {
              V_master[iLayer] = rearrange(v_master[iLayer], rearrangeBlock[iLayer]);
              cV_master[iLayer] = fft(V_master[iLayer], dimCount - 1, plan_v[iLayer]);
              hipStreamSynchronize(0);
            }
            #pragma omp barrier
            cV[iLayer] = cV_master[iLayer];
          } else if (rLayerCount) {
            #pragma omp master
            thrust::copy(v_master[iLayer].begin(), v_master[iLayer].end(), v_flat[0].begin());
          }
          hipStreamSynchronize(0);
          #pragma omp barrier
        }
      }

      #pragma omp master
      {
        V_master[0] = (V_master[0] * dbm.getStddev() + dbm.getMean()) * repeat(hMask[0], visSize[0] / layerSize[0]);
        v_master[0] = rearrange_r(V_master[0], rearrangeBlock[0]);
        outputs->push_back(boost::make_shared<host_tensor_t>(v_master[0]));
        hipStreamSynchronize(0);
      }
      #pragma omp barrier

      #pragma omp master
      if (monitor)
        monitor->reportProgress(100. * (iSample + 1) / getSampleCount());
    } /* end of samples */

    hipStreamSynchronize(0);
    #pragma omp barrier

    // Free up memory
    for (size_t iLayer = 0; iLayer < cLayerCount; ++iLayer) {
      for (size_t k = tid; k < cF[iLayer].size(); k += gpuCount) {
        cF[iLayer][k] = cc[iLayer][k] = boost::shared_ptr<ctensor_t>();
      }
    }

    if (tid == 0) {
      for (int i = 1; i < gpuCount; ++i)
        hipDeviceDisablePeerAccess(i);
    } else {
      hipDeviceDisablePeerAccess(0);
    }

  } /* end of parallel */

  newState->setSamples(outputs);
}

}

}


