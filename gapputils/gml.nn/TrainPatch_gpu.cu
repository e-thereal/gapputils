#include "hip/hip_runtime.h"
/*
 * TrainPatch_gpu.cu
 *
 *  Created on: Aug 14, 2014
 *      Author: tombr
 */

#include "TrainPatch.h"

#include <tbblas/deeplearn/nn.hpp>
#include <tbblas/io.hpp>
#include <tbblas/sum.hpp>
#include <tbblas/dot.hpp>
#include <tbblas/reshape.hpp>
#include <tbblas/ones.hpp>

#include <tbblas/sequence_iterator.hpp>

namespace gml {

namespace nn {

TrainPatchChecker::TrainPatchChecker() {
  TrainPatch test;
  test.initializeClass();

  CHECK_MEMORY_LAYOUT2(InitialModel, test);
  CHECK_MEMORY_LAYOUT2(TrainingSet, test);
  CHECK_MEMORY_LAYOUT2(Labels, test);
  CHECK_MEMORY_LAYOUT2(Mask, test);
  CHECK_MEMORY_LAYOUT2(PatchWidth, test);
  CHECK_MEMORY_LAYOUT2(PatchHeight, test);
  CHECK_MEMORY_LAYOUT2(PatchDepth, test);
  CHECK_MEMORY_LAYOUT2(PatchCount, test);
  CHECK_MEMORY_LAYOUT2(SelectionMethod, test);
  CHECK_MEMORY_LAYOUT2(PositiveRatio, test);
  CHECK_MEMORY_LAYOUT2(MinimumBucketSizes, test);
  CHECK_MEMORY_LAYOUT2(BucketRatio, test);

  CHECK_MEMORY_LAYOUT2(EpochCount, test);
  CHECK_MEMORY_LAYOUT2(BatchSize, test);
  CHECK_MEMORY_LAYOUT2(Objective, test);
  CHECK_MEMORY_LAYOUT2(SensitivityRatio, test);
  CHECK_MEMORY_LAYOUT2(Method, test);
  CHECK_MEMORY_LAYOUT2(LearningRate, test);
  CHECK_MEMORY_LAYOUT2(WeightCosts, test);
  CHECK_MEMORY_LAYOUT2(DropoutRates, test);
  CHECK_MEMORY_LAYOUT2(RandomizeTraining, test);
  CHECK_MEMORY_LAYOUT2(Model, test);
  CHECK_MEMORY_LAYOUT2(Patches, test);
  CHECK_MEMORY_LAYOUT2(Targets, test);
  CHECK_MEMORY_LAYOUT2(Predictions, test);
}

void TrainPatch::update(IProgressMonitor* monitor) const {
  using namespace tbblas;

  Logbook& dlog = getLogbook();
  dlog.setSeverity(Severity::Message);

  const int dimCount = host_tensor_t::dimCount;

  typedef int8_t bucket_id_t;
  typedef model_t::value_t value_t;
  typedef tbblas::tensor<value_t, 2, true> matrix_t;
  typedef tbblas::tensor<value_t, 2, false> host_matrix_t;

  typedef tbblas::tensor<value_t, dimCount, true> tensor_t;
  typedef tbblas::tensor<bucket_id_t, dimCount, false> bucket_id_tensor_t;
  typedef tensor_t::dim_t dim_t;

  if (getTrainingSet()->size() != getLabels()->size()) {
    dlog(Severity::Warning) << "The sizes of the training and label set don't match. Aborting!";
    return;
  }

  if (getSelectionMethod() == PatchSelectionMethod::LeitnerSystem && getPositiveRatio() <= 0) {
    dlog(Severity::Warning) << "The positive ratio must be greater than 0 to use the Leitner system. Aborting!";
    return;
  }

  if (getSelectionMethod() == PatchSelectionMethod::LeitnerSystem && _MinimumBucketSizes.size() < 2) {
    dlog(Severity::Warning) << "You need at least 2 buckets in order to use the Leitner system. Aborting!";
    return;
  }

  boost::shared_ptr<model_t> model(new model_t(*getInitialModel()));

  tbblas::deeplearn::nn<value_t> nn(*model);
  nn.set_objective_function(getObjective());
  nn.set_sensitivity_ratio(getSensitivityRatio());

  for (size_t i = 0; i < model->layers().size() && i < _DropoutRates.size(); ++i) {
    nn.set_dropout_rate(i, _DropoutRates[i]);
  }

  nn.visibles().resize(seq((int)getBatchSize() * getPatchCount(), (int)model->visibles_count()));

  // Prepare data
  v_host_tensor_t& data = *getTrainingSet();
  v_host_tensor_t& labels = *getLabels();

  if (getPatchWidth() * getPatchHeight() * getPatchDepth() * data[0]->size()[dimCount - 1] != model->visibles_count()) {
    dlog(Severity::Warning) << "Patch dimension doesn't match the number of visible units of the neural network. Aborting!";
    return;
  }

  if (getPositiveRatio() >= 0.0 && labels[0]->size()[dimCount - 1] != 1) {
    dlog(Severity::Warning) << "Positive ratio can only be used for binary classification (channels of the label image must be 1). Aborting!";
    return;
  }

  dim_t patchSize = seq(getPatchWidth(), getPatchHeight(), getPatchDepth(), data[0]->size()[dimCount - 1]);
  dim_t labelSize = seq(1, 1, 1, labels[0]->size()[dimCount - 1]);
  dim_t range = data[0]->size() - patchSize + 1;
  dim_t patchCenter = patchSize / 2 * seq(1,1,1,0);

  matrix_t yBatch(getBatchSize() * getPatchCount(), model->hiddens_count());
  host_matrix_t h_predictions(yBatch.size());

  matrix_t res;
  value_t weightcost = getWeightCosts();
  value_t initialmomentum = 0.5f;
  value_t finalmomentum = 0.9f;
  value_t momentum;

  const int batchSize = getBatchSize();
  const int batchCount = data.size() / batchSize;

  value_t error, PPV, DSC, TPR, TNR;
  tensor_t tensor, label;
  host_tensor_t h_label;

  boost::shared_ptr<v_host_tensor_t> patches(new v_host_tensor_t());
  newState->setPatches(patches);

  boost::shared_ptr<v_host_tensor_t> targets(new v_host_tensor_t());
  newState->setTargets(targets);

  boost::shared_ptr<v_host_tensor_t> predictions(new v_host_tensor_t());
  newState->setPredictions(predictions);

  // bucket IDs
  // -1: ignore
  //  0: unknown
  //  1: Bucket one (hard cases)
  //  2: Bucket two (correctly solved once)
  //  3: Bucket three (correctly solved twice)
  //  4: Bucket four (correctly solved four times)

  // At the beginning, bucket one will always be refilled with random cases to contain at least the minimum number of samples
  // This is temporarily. Filled samples are not tagged as belonging to bucket one.

  std::vector<bucket_id_tensor_t> bucketIds;

  std::vector<dim_t> positiveLocations, negativeLocations, maskLocations, selectedLocations(getPatchCount() * getBatchSize());
  std::vector<std::vector<dim_t> > bucketLocations(_MinimumBucketSizes.size());
  std::vector<size_t> selectedSamples(getBatchSize());

  if (getSelectionMethod() == PatchSelectionMethod::LeitnerSystem) {
    bucketIds.resize(labels.size());

    if (getMask()) {
      host_tensor_t& mask = *getMask();
      for (size_t iSample = 0; iSample < bucketIds.size(); ++iSample) {
        host_tensor_t& label = *labels[iSample];
        bucketIds[iSample] = -1 * ones<value_t>(label.size()) + (mask > 0);
      }
    } else {
      for (size_t iSample = 0; iSample < bucketIds.size(); ++iSample) {
        host_tensor_t& label = *labels[iSample];
        bucketIds[iSample] = zeros<bucket_id_t>(label.size());
      }
    }
  } else {
    if (getMask()) {
      host_tensor_t& mask = *getMask();
      for (sequence_iterator<dim_t> pos(seq<dimCount>(0), (mask.size() - patchSize) * seq(1, 1, 1, 0) + seq(0, 0, 0, 1)); pos; ++pos) {
        if (mask[*pos + patchSize / 2 * seq(1, 1, 1, 0)] > 0) {   // The center is within the lesion mask
          maskLocations.push_back(*pos);
        }
      }
    }
  }

  dlog() << "Preparation finished. Starting training.";

  for (int iEpoch = 0; iEpoch < getEpochCount() && (monitor ? !monitor->getAbortRequested() : true); ++iEpoch) {

    PPV = DSC = TPR = TNR = error = 0;

    if (iEpoch < 10)
      momentum = initialmomentum;
    else
      momentum = finalmomentum;

    for (int iBatch = 0; iBatch < batchCount; ++iBatch) {

      for (int iSample = 0; iSample < batchSize; ++iSample) {
        // Fill batch with random patches
        tensor = *data[iBatch * batchSize + iSample];
        label = *labels[iBatch * batchSize + iSample];
        h_label = *labels[iBatch * batchSize + iSample];

        selectedSamples[iSample] = iBatch * batchSize + iSample;

        int lesionPatchCount = 0;

        if (getSelectionMethod() == PatchSelectionMethod::LeitnerSystem) {
          positiveLocations.clear();
          negativeLocations.clear();

          for (size_t iBucket = 0; iBucket < bucketLocations.size(); ++iBucket)
            bucketLocations[iBucket].clear();

          // Fill buckets with potential locations
          // Also selected unknown positive and negative samples for quicker refilling the first bucket if needed

          for (sequence_iterator<dim_t> pos(seq<dimCount>(0), (label.size() - patchSize) * seq(1, 1, 1, 0) + seq(0, 0, 0, 1)); pos; ++pos) {
            const dim_t center = *pos + patchSize / 2 * seq(1, 1, 1, 0);  // center of the patch
            const bucket_id_t bucketId = bucketIds[iSample][center];

            if (bucketId > 0) {
              bucketLocations[bucketId - 1].push_back(*pos);
            } else if (bucketId == 0) {
              if (h_label[center] > 0) {    // The center is within a lesion
                positiveLocations.push_back(*pos);
              } else {
                negativeLocations.push_back(*pos);
              }
            }
          }

          while (bucketLocations[0].size() < _MinimumBucketSizes[0]) {
            if ((float)rand() / (float)RAND_MAX < getPositiveRatio()) {
              bucketLocations[0].push_back(positiveLocations[rand() % positiveLocations.size()]);
            } else {
              bucketLocations[0].push_back(negativeLocations[rand() % negativeLocations.size()]);
            }
          }

          if (iSample == 0 && iBatch == 0) {
            std::cout << "Buckets: " << bucketLocations[0].size();
            for (size_t i = 1; i < bucketLocations.size(); ++i)
              std::cout << ", " << bucketLocations[i].size();
            std::cout << std::endl;
          }

        } else {
          if (getPositiveRatio() >= 0) {
            positiveLocations.clear();
            for (sequence_iterator<dim_t> pos(seq<dimCount>(0), (label.size() - patchSize) * seq(1, 1, 1, 0) + seq(0, 0, 0, 1)); pos; ++pos) {
              if (h_label[*pos + patchSize / 2 * seq(1, 1, 1, 0)] > 0) {   // The center is within a lesion
                positiveLocations.push_back(*pos);
              }
            }
          }
        }

        for (int iPatch = 0; iPatch < getPatchCount(); ++iPatch) {
          dim_t topleft;

          if (getSelectionMethod() == PatchSelectionMethod::LeitnerSystem) {

            // Initialize the topleft with a patch location from the first bucket
            topleft = bucketLocations[0][rand() % bucketLocations[0].size()];

            // If we don't want a sample from the first bucket, try the next buckets
            // If a randomly selected bucket doesn't fulfil the size requirements, fall back to the first bucket (keep initial value)
            // If no bucket was randomly selected, also fall back to the first bucket (keep initial value)
            if ((float)rand() / (float)RAND_MAX > getBucketRatio()) {
              for (size_t iBucket = 1; iBucket < bucketLocations.size(); ++iBucket) {
                if ((float)rand() / (float)RAND_MAX < getBucketRatio()) {
                  if (bucketLocations[iBucket].size() >= _MinimumBucketSizes[iBucket])
                    topleft = bucketLocations[iBucket][rand() % bucketLocations[iBucket].size()];
                  break;
                }
              }
            }

            selectedLocations[iSample * getPatchCount() + iPatch] = topleft;
          } else {
            if (positiveLocations.size() && (float)rand() / (float)RAND_MAX < getPositiveRatio())
              topleft = positiveLocations[rand() % positiveLocations.size()];
            else if (maskLocations.size())
              topleft = maskLocations[rand() % maskLocations.size()];
            else
              topleft = seq(rand() % range[0], rand() % range[1], rand() % range[2], 0);
          }

          if (h_label[topleft + patchCenter] > 0.1)
            ++lesionPatchCount;

          // Fill visible units and targets
          row(nn.visibles(), iSample * getPatchCount() + iPatch) = reshape(tensor[topleft, patchSize], 1, model->visibles_count());
          row(yBatch, iSample * getPatchCount() + iPatch) = reshape(label[topleft + patchCenter, labelSize], 1, model->hiddens_count());

          if (iEpoch + 1 == getEpochCount() && iBatch + 1 == batchCount && iSample + 1 == batchSize) {
            patches->push_back(boost::make_shared<host_tensor_t>(tensor[topleft, patchSize]));
            targets->push_back(boost::make_shared<host_tensor_t>(label[topleft + patchCenter, labelSize]));
          }
        }
        if (iEpoch == 0 && iBatch == 0 && iSample == 0) {
          dlog(Severity::Trace) << "Lesion patches = " << lesionPatchCount << "; total number of patches = " << getPatchCount();
        }
      }

      // Perform forward propagation
      nn.normalize_visibles();

      // Update model
      switch (getMethod()) {
      case TrainingMethod::Momentum:
        nn.momentum_update(yBatch, getLearningRate(), momentum, weightcost);
        break;

      case TrainingMethod::AdaDelta:
        nn.adadelta_update(yBatch, getLearningRate(), momentum, weightcost);
        break;
      }

      // Calculate errors
      error += sqrt(dot(nn.hiddens() - yBatch, nn.hiddens() - yBatch) / yBatch.size()[0]);

      if (iEpoch + 1 == getEpochCount() && iBatch + 1 == batchCount) {
        for (size_t iPatch = 0; iPatch < getPatchCount(); ++iPatch)
          predictions->push_back(boost::make_shared<host_tensor_t>(reshape(row(nn.hiddens(), (batchSize - 1) * getPatchCount() + iPatch), labelSize)));
      }

      PPV += sum((yBatch > 0.5) * (nn.hiddens() > 0.5)) / sum(nn.hiddens() > 0.5);
      DSC += 2 * sum ((yBatch > 0.5) * (nn.hiddens() > 0.5)) / (sum(yBatch > 0.5) + sum(nn.hiddens() > 0.5));
      TPR += sum((yBatch > 0.5) * (nn.hiddens() > 0.5)) / sum(yBatch > 0.5);
      TNR += sum((yBatch < 0.5) * (nn.hiddens() < 0.5)) / sum(yBatch < 0.5);

      if (getSelectionMethod() == PatchSelectionMethod::LeitnerSystem) {
        // Figure out if patches were classified correctly and update the bucketIds
        h_predictions = nn.hiddens();

        for (size_t iPatch = 0; iPatch < selectedLocations.size(); ++iPatch) {
          const dim_t center = selectedLocations[iPatch] + patchSize / 2 * seq(1, 1, 1, 0);  // center of the patch
          const size_t iSample = selectedSamples[iPatch / getPatchCount()];
          if (h_predictions.data()[iPatch] > 0.5) {
            ++bucketIds[iSample][center];
          } else {
            bucketIds[iSample][center] = 1;
          }
        }
      }
    }

    dlog(Severity::Trace) << "Error at epoch " << iEpoch + 1 << " of " << getEpochCount() << " epochs: " << error / batchCount
        << " (PPV = " << PPV / batchCount << ", DSC = " << DSC / batchCount << ", TPR = " << TPR / batchCount << ", TNR = " << TNR / batchCount << ")";

    if (monitor)
      monitor->reportProgress(100. * (iEpoch + 1) / getEpochCount());
  }

  newState->setModel(model);
}

}

}
